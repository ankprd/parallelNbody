#include "hip/hip_runtime.h"
#include <string.h>
#include <stdio.h>
#include <stdlib.h>
#include "nbody.h"

// CUDA runtime
#include <hip/hip_runtime.h>

// includes
#include <helper_functions.h> // helper for shared functions common to CUDA Samples
#include <hip/hip_runtime_api.h>	  // helper functions for CUDA error checking and initialization

extern "C"
{
#include <hip/hip_runtime.h>
}
#define MEMSIZE 30

//int nparticles;
extern particle_t *particles;

particle_t  *d_particles, *d_nparticles;

__device__ void compute_force(particle_t*p, double x_pos, double y_pos, double mass) {
  double x_sep, y_sep, dist_sq, grav_base;

  x_sep = x_pos - p->x_pos;
  y_sep = y_pos - p->y_pos;
  dist_sq = MAX((x_sep*x_sep) + (y_sep*y_sep), 0.01);

  /* Use the 2-dimensional gravity rule: F = d * (GMm/d^2) */
  grav_base = GRAV_CONSTANT*(p->mass)*(mass)/dist_sq;

  p->x_force += grav_base*x_sep;
  p->y_force += grav_base*y_sep;
}

__global__ void calcForce(particle_t *d_particles, particle_t *d_nparticles, int d_nbP)
{
  int i, j;
  i = blockIdx.x * blockDim.x + threadIdx.x;
  if (i < d_nbP){
    d_nparticles[i].x_force = 0;
    d_nparticles[i].y_force = 0;
    //printf("calculating for part %d \n", i);
    for(j = 0; j < d_nbP; j++)
        compute_force(&d_nparticles[i], d_particles[j].x_pos, d_particles[j].y_pos, d_particles[j].mass);//on modifie nparticle, et on prend les infos de d_particle
  }
}

extern "C" void initCuda(){
  hipMalloc((void**)&d_particles, nparticles * sizeof(particle_t));
  hipMalloc((void**)&d_nparticles, nparticles * sizeof(particle_t));
}

extern "C" void finalizeCuda(){
  hipFree(d_particles);
  hipFree(d_nparticles);
}

extern "C" void all_move_particles(double step)
{
  //nparticles = nbParts;
  //printf("Nb parts in gpu : %d\n", nparticles);
  int i;
  /*for(i=0; i<nparticles; i++) {
    particle_t*p = &particles[i];
    printf("in .cu particle={pos=(%f,%f), vel=(%f,%f)}\n", p->x_pos, p->y_pos, p->x_vel, p->y_vel);
  }*/

  hipMemcpy(d_particles, particles, nparticles * sizeof(particle_t), hipMemcpyHostToDevice);
  hipMemcpy(d_nparticles, particles, nparticles * sizeof(particle_t), hipMemcpyHostToDevice);

  calcForce<<<1000000, 10>>>(d_particles, d_nparticles, nparticles);

  /*for(i=0; i<nparticles; i++) {
    particle_t*p = &particles[i];
    printf("in .cu particle={pos=(%f,%f), vel=(%f,%f), force=(%f,%f)}\n", p->x_pos, p->y_pos, p->x_vel, p->y_vel, p->x_force, p->y_force);
  }*/

  hipMemcpy(particles, d_nparticles, nparticles * sizeof(particle_t), hipMemcpyDeviceToHost);
  /*for(i=0; i<nparticles; i++) {
    particle_t*p = &particles[i];
    printf("in .cu after calc particle={pos=(%f,%f), vel=(%f,%f)}\n", p->x_pos, p->y_pos, p->x_vel, p->y_vel);
  }*/
}

