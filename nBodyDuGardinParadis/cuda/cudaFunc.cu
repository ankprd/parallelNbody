#include "hip/hip_runtime.h"
/*
** cudaFunc.cu - helper functions for cudaBruteforce.c
**
**/

#include <string.h>
#include <stdio.h>
#include <stdlib.h>
#include "nbody.h"

// CUDA runtime
#include <hip/hip_runtime.h>

extern "C"
{
#include <hip/hip_runtime.h>
}
#define MEMSIZE 30

//int nparticles;
extern particle_t *particles;

particle_t  *d_particles, *d_nparticles;

__device__ void compute_force(particle_t*p, double x_pos, double y_pos, double mass) {
  double x_sep, y_sep, dist_sq, grav_base;

  x_sep = x_pos - p->x_pos;
  y_sep = y_pos - p->y_pos;
  dist_sq = MAX((x_sep*x_sep) + (y_sep*y_sep), 0.01);

  /* Use the 2-dimensional gravity rule: F = d * (GMm/d^2) */
  grav_base = GRAV_CONSTANT*(p->mass)*(mass)/dist_sq;

  p->x_force += grav_base*x_sep;
  p->y_force += grav_base*y_sep;
}

__global__ void calcForce(particle_t *d_particles, particle_t *d_nparticles, int d_nbP)
{
  int i, j;
  i = blockIdx.x * blockDim.x + threadIdx.x;
  if (i < d_nbP){
    d_nparticles[i].x_force = 0;
    d_nparticles[i].y_force = 0;
    //printf("calculating for part %d \n", i);
    for(j = 0; j < d_nbP; j++)
        compute_force(&d_nparticles[i], d_particles[j].x_pos, d_particles[j].y_pos, d_particles[j].mass);//on modifie nparticle, et on prend les infos de d_particle
  }
}

extern "C" void initCuda(){
  hipMalloc((void**)&d_particles, nparticles * sizeof(particle_t));
  hipMalloc((void**)&d_nparticles, nparticles * sizeof(particle_t));
}

extern "C" void finalizeCuda(){
  hipFree(d_particles);
  hipFree(d_nparticles);
}

extern "C" void all_move_particles(double step)
{
  hipMemcpy(d_particles, particles, nparticles * sizeof(particle_t), hipMemcpyHostToDevice);
  hipMemcpy(d_nparticles, particles, nparticles * sizeof(particle_t), hipMemcpyHostToDevice);

  calcForce<<<1000000, 10>>>(d_particles, d_nparticles, nparticles);

  hipMemcpy(particles, d_nparticles, nparticles * sizeof(particle_t), hipMemcpyDeviceToHost);
}

